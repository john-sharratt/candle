#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"
#include<stdint.h>

// Optimized kernel for f32 - uses atomic operations for thread safety
extern "C" __global__ void sub_at_indices_f32(
    float* data,
    const uint32_t* indices,
    const size_t num_indices,
    const float value
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < num_indices) {
        const uint32_t token_id = indices[idx];
        atomicAdd(&data[token_id], -value);
    }
}

// Optimized kernel for f16
extern "C" __global__ void sub_at_indices_f16(
    __half* data,
    const uint32_t* indices,
    const size_t num_indices,
    const float value
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < num_indices) {
        const uint32_t token_id = indices[idx];
        const __half val = __float2half(value);
        
        #if __CUDA_ARCH__ >= 700
        // Use native half atomics on newer GPUs
        atomicAdd(&data[token_id], __hneg(val));
        #else
        // Fallback using CAS (Compare-And-Swap) for thread safety
        unsigned short int* address_as_us = (unsigned short int*)&data[token_id];
        unsigned short int old = *address_as_us;
        unsigned short int assumed;
        
        do {
            assumed = old;
            __half old_half = __ushort_as_half(assumed);
            __half new_half = __float2half(__half2float(old_half) - value);
            old = atomicCAS(address_as_us, assumed, __half_as_ushort(new_half));
        } while (assumed != old);
        #endif
    }
}

// Optimized kernel for bf16
extern "C" __global__ void sub_at_indices_bf16(
    __hip_bfloat16* data,
    const uint32_t* indices,
    const size_t num_indices,
    const float value
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < num_indices) {
        const uint32_t token_id = indices[idx];
        
        #if __CUDA_ARCH__ >= 800
        // Use native bfloat16 atomics on Ampere and newer
        const __hip_bfloat16 val = __float2bfloat16(value);
        atomicAdd(&data[token_id], __hneg(val));
        #else
        // Fallback using CAS for thread safety
        unsigned short int* address_as_us = (unsigned short int*)&data[token_id];
        unsigned short int old = *address_as_us;
        unsigned short int assumed;
        
        do {
            assumed = old;
            __hip_bfloat16 old_bf16 = *reinterpret_cast<__hip_bfloat16*>(&assumed);
            float old_float = __bfloat162float(old_bf16);
            __hip_bfloat16 new_bf16 = __float2bfloat16(old_float - value);
            old = atomicCAS(address_as_us, assumed, *reinterpret_cast<unsigned short int*>(&new_bf16));
        } while (assumed != old);
        #endif
    }
}

// Optimized kernel for f64
extern "C" __global__ void sub_at_indices_f64(
    double* data,
    const uint32_t* indices,
    const size_t num_indices,
    const double value
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < num_indices) {
        const uint32_t token_id = indices[idx];
        atomicAdd(&data[token_id], -value);
    }
}
